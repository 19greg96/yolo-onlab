#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

void forward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if (!state.train) return;
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */
	printf("forward dropout: size: %d, inputs: %d, outputs: %d, batch: %d\n", size, layer.inputs, layer.outputs, layer.batch);
    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, size, layer.rand_gpu, layer.probability, layer.scale);
    CHECK_CUDA(hipPeekAtLastError());
	
	// this is done in parser.c
	// layer.output_gpu = state.input; // copy pointers, because in network_kernels.cu
	// we do the following after every forward step:
	//state.input = l.output_gpu;
}

void backward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if(!state.delta) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.delta, size, layer.rand_gpu, layer.probability, layer.scale);
    CHECK_CUDA(hipPeekAtLastError());
}
